#include <iostream>
#include <fstream>
#include <vector>
#include <limits.h>
#include <hip/hip_runtime.h>

#define INF INT_MAX

__global__ void bellmanFordKernel(int *d_edges, int *d_weights, int *d_distance, int numVertices, int numEdges) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < numEdges) {
        int u = d_edges[tid * 2];
        int v = d_edges[tid * 2 + 1];
        int weight = d_weights[tid];
        if (d_distance[u] != INF && d_distance[u] + weight < d_distance[v]) {
            d_distance[v] = d_distance[u] + weight;
        }
    }
}

int main() {
    std::vector<int> h_edges; // Use a single vector to store both u and v of each edge
    std::vector<int> h_weights;
    int V = 0; // Number of vertices
    int E = 0; // Count edges while reading

    // Read graph from file
    std::ifstream graphFile("large_graph.txt");
    int u, v, w;
    while (graphFile >> u >> v >> w) {
        h_edges.push_back(u);
        h_edges.push_back(v);
        h_weights.push_back(w);
        int maxVertex = std::max(u, v);
        V = std::max(V, maxVertex + 1);
        E++;
    }
    graphFile.close();

    // Allocate memory on host
    int *h_distance = new int[V];
    for (int i = 0; i < V; ++i) {
        h_distance[i] = INF;
    }
    h_distance[0] = 0; // Assuming source vertex is 0

    // Allocate memory on device
    int *d_edges, *d_weights, *d_distance;
    hipMalloc(&d_edges, sizeof(int) * 2 * E);
    hipMalloc(&d_weights, sizeof(int) * E);
    hipMalloc(&d_distance, sizeof(int) * V);

    // Copy data from host to device
    hipMemcpy(d_edges, h_edges.data(), sizeof(int) * 2 * E, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, h_weights.data(), sizeof(int) * E, hipMemcpyHostToDevice);
    hipMemcpy(d_distance, h_distance, sizeof(int) * V, hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 block(256);
    dim3 grid((E + block.x - 1) / block.x);

    // Execute the Bellman-Ford algorithm
    for (int i = 0; i < V - 1; ++i) {
        bellmanFordKernel<<<grid, block>>>(d_edges, d_weights, d_distance, V, E);
        hipDeviceSynchronize();
    }

    // Copy results back to host
    hipMemcpy(h_distance, d_distance, sizeof(int) * V, hipMemcpyDeviceToHost);

    // Print the shortest distances
    std::cout << "Vertex Distance from Source" << std::endl;
    for (int i = 0; i < V; ++i) {
        std::cout << i << "\t\t" << (h_distance[i] == INF ? "INF" : std::to_string(h_distance[i])) << std::endl;
    }

    // Free device memory
    hipFree(d_edges);
    hipFree(d_weights);
    hipFree(d_distance);

    // Free host memory
    delete[] h_distance;

    return 0;
}
